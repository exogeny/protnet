#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::Half, complex_t>(SSMParamsBase &params, hipStream_t stream);
