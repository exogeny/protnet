#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
